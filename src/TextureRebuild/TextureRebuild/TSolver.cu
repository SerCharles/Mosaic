#include "hip/hip_runtime.h"
/*
filename: TSolver.cu
description: used in Gauss-Newton method
date: 4/25/2019
*/


//system requirements
#include <time.h>
#include <iostream>
#include <fstream>
#include <windows.h>
#include <vector>
#include <algorithm>
using namespace std;

//cv requirements
#include <opencv2/opencv.hpp>

//algebra requirements
#include <Eigen/Eigen>
#include <Eigen/Sparse>
#include <Eigen/SparseCholesky>
#include <limits>

//personal requirements
#include "TDefine.hpp"
#include "TInput.h"
#include "TSolver.cuh"

//cuda
#include <hipsolver.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

void TSolver::m_GaussNewton()
{
	int total_time = 0;
	m_GetFX_R();
	while (1)
	{
		double old_answer = m_Answer_R;
		m_GetAnswer_R();
		if (m_Answer_R >= old_answer) break;
		m_GetJacobi_R();
		m_PrepareSolve_R();
		//m_Test();
		double max = m_Solve_R();
		total_time++;
		if (max <= m_MaxDifference || total_time >= m_MaxTime)
		{
			break;
		}
		else
		{
			for (int i = 0; i < m_Size; i++)
			{
				m_X_R[i] += m_DX_R[i];
			}
		}
	}
	total_time = 0;
}

void TSolver::m_RenewColor()
{
	for (int i = 0; i < g_PointList.size(); i++)
	{
		for (int j = 0; j < g_PointList[i].m_GroupList.size(); j++)
		{
			int group = g_PointList[i].m_GroupList[j];
			int new_color_r = m_X_R[i * g_GroupList.size() + group];

			///todo: g and b
			//int new_color_g = m_X_G.coeff(i * g_GroupList.size() + group, 0);
			//int new_color_b = m_X_B.coeff(i * g_GroupList.size() + group, 0);
			Eigen::Vector3i new_color(new_color_r, 0, 0);
			g_PointList[i].m_ColorList[j] += new_color;
		}
	}
}

bool JudgeAdjacent(int i1, int i2)
{
	for (int i = 0; i < g_PointList[i1].m_PointLink.size(); i++)
	{
		if (i2 == g_PointList[i1].m_PointLink[i]) return 1;
	}
	return 0;
}

void TSolver::m_GetFX_R()
{
	memset(m_FX_R, 0, m_Size * sizeof(int));

	for (int i = 0; i < g_PointList.size(); i++)
	{
		for (int j = 0; j < g_PointList[i].m_GroupList.size(); j++)
		{
			int num = i * g_GroupList.size() + g_PointList[i].m_GroupList[j];
			double r = g_PointList[i].m_ColorList[j](0);
			m_FX_R[num] = r;
		}
	}
}

void TSolver::m_GetAnswer_R()
{
	double ans = 0;
	for (int j = 0; j < g_GroupList.size(); j++)
	{
		for (int i1 = 0; i1 < g_GroupList[j].m_PointList.size() - 1; i1++)
		{
			for (int i2 = i1 + 1; i2 < g_GroupList[j].m_PointList.size(); i2++)
			{
				int point_1 = g_GroupList[j].m_PointList[i1];
				int point_2 = g_GroupList[j].m_PointList[i2];
				if (JudgeAdjacent(point_1, point_2))
				{
					double temp_1 = m_X_R[point_1 * g_GroupList.size() + j];
					double temp_2 = m_X_R[point_2 * g_GroupList.size() + j];
					ans += (temp_1 - temp_2)*(temp_1 - temp_2);
				}
			}
		}
	}
	for (int i = 0; i < g_PointList.size(); i++)
	{
		for (int j1 = 0; j1 < g_PointList[i].m_GroupList.size() - 1; j1++)
		{
			for (int j2 = j1 + 1; j2 < g_PointList[i].m_GroupList.size(); j2++)
			{
				int group_1 = g_PointList[i].m_GroupList[j1];
				int group_2 = g_PointList[i].m_GroupList[j2];
				double temp_g_1 = m_X_R[i * g_GroupList.size() + group_1];
				double temp_f_1 = m_FX_R[i * g_GroupList.size() + group_1];
				double temp_g_2 = m_X_R[i * g_GroupList.size() + group_2];
				double temp_f_2 = m_FX_R[i * g_GroupList.size() + group_2];
				ans += m_Lambda *
					(temp_g_1 + temp_f_1 - temp_f_2 - temp_g_2) * (temp_g_1 + temp_f_1 - temp_f_2 - temp_g_2);
			}
		}
	}
	m_Answer_R = sqrt(ans);
}

void TSolver::m_GetJacobi_R()
{
	m_NonZeroJacobi_R.clear();
	//m_JacobiNorm = 0;
	memset(m_Jacobi_R, 0, m_Size * sizeof(int));
	for (int i = 0; i < g_PointList.size(); i++)
	{
		for (int j = 0; j < g_PointList[i].m_GroupList.size(); j++)
		{
			int group = g_PointList[i].m_GroupList[j];
			double ans_r_1 = 0;
			double ans_r_2 = 0;
			double num_r = m_X_R[i * g_GroupList.size() + group];
			double f_r = m_FX_R[i * g_GroupList.size() + group];
			for (int k = 0; k < g_GroupList[group].m_PointList.size(); k++)
			{
				int point = g_GroupList[group].m_PointList[k];
				if (JudgeAdjacent(i, point))
				{
					ans_r_1 += num_r;
					ans_r_1 -= m_X_R[point * g_GroupList.size() + group];
				}
			}
			for (int k = 0; k < g_PointList[i].m_GroupList.size(); k++)
			{
				int group_nova = g_PointList[i].m_GroupList[k];
				ans_r_2 += num_r; ans_r_2 += f_r;

				ans_r_2 -= m_X_R[i * g_GroupList.size() + group_nova];
				ans_r_2 -= m_FX_R[i * g_GroupList.size() + group_nova];
			}
			double ans_r = (ans_r_1 + m_Lambda * ans_r_2) * 2;

			m_Jacobi_R[i * g_GroupList.size() + group] = ans_r / m_Answer_R;
		}
	}
	for (int i = 0; i < m_Size; i++)
	{
		if (m_Jacobi_R[i] != 0)
		{
			m_NonZeroJacobi_R.push_back(i);
		}
	}



}

void TSolver::m_PrepareSolve_R()
{
	//get b
	for (int i = 0; i < m_Size; i++)
	{
		m_B_R[i] = - m_Jacobi_R[i] * m_Answer_R;
	}



	//get H(x)
	//Actually it is Q(x)=H(x)*J(x).norm2
	m_Hermit_R.m_Renew();
	m_Hermit_R.m_NonZeros = m_NonZeroJacobi_R.size() *  m_NonZeroJacobi_R.size() + 
		m_Size - m_NonZeroJacobi_R.size();

	m_Hermit_R.m_Place_x = new int[m_Hermit_R.m_NonZeros];
	m_Hermit_R.m_Place_y = new int[m_Hermit_R.m_NonZeros];
	m_Hermit_R.m_Data = new double[m_Hermit_R.m_NonZeros];
	int flag = 0;
	for (int i = 0; i < m_NonZeroJacobi_R.size(); i++)
	{
		int start = 0, end = 0;
		if (i == 0)
		{
			start = 0;
			end = m_NonZeroJacobi_R[0];
		}
		else
		{
			start = m_NonZeroJacobi_R[i - 1] + 1;
			end = m_NonZeroJacobi_R[i];
		}
		for (int j = start; j < end; j++)
		{
			m_Hermit_R.m_Place_x[flag] = j;
			m_Hermit_R.m_Place_y[flag] = j;
			m_Hermit_R.m_Data[flag] = m_HermitDiag;
			flag++;
		}
		for (int jj = 0; jj < m_NonZeroJacobi_R.size(); jj++)
		{
			m_Hermit_R.m_Place_x[flag] = m_NonZeroJacobi_R[i];
			m_Hermit_R.m_Place_y[flag] = m_NonZeroJacobi_R[jj];
			double data = m_Jacobi_R[m_NonZeroJacobi_R[i]] * m_Jacobi_R[m_NonZeroJacobi_R[jj]];
			if (i == jj)
			{
				//��ӶԽ������Ʋ�������֤����
				data += m_HermitDiag;
			}
			m_Hermit_R.m_Data[flag] = data;
			flag++;
		}
	}
	for (int j = m_NonZeroJacobi_R[m_NonZeroJacobi_R.size() - 1] + 1; j < m_Size; j++)
	{
		m_Hermit_R.m_Place_x[flag] = j;
		m_Hermit_R.m_Place_y[flag] = j;
		m_Hermit_R.m_Data[flag] = m_HermitDiag;
		flag++;
	}


}

void TSolver::m_Test()
{
	ofstream file_1("hermit.txt");
	file_1 << m_Hermit_R.m_NonZeros << endl;
	for (int i = 0; i <= m_Size; i++) file_1 << m_Hermit_R.m_Place_x[i]<<" ";
	file_1 << endl;
	for (int i = 0; i < m_Hermit_R.m_NonZeros; i++) file_1 << m_Hermit_R.m_Place_y[i]<<" ";
	file_1 << endl;
	for(int i = 0; i < m_Hermit_R.m_NonZeros;i++) file_1 << m_Hermit_R.m_Data[i] << " ";
	file_1 << endl;
	ofstream file_2("b.txt");
	for (int i = 0; i < m_Size; i++) file_2 << m_B_R[i] << " ";
	ofstream file_3("jacobi.txt");
	for (int i = 0; i < m_Size; i++) file_3 << m_Jacobi_R[i] << " ";

}

double TSolver::m_Solve_R()
{
	memset(m_DX_R, 0, m_Size * sizeof(int));

	//define status and variables
	hipsparseStatus_t status;
	hipblasStatus_t status_cub;
	hipsparseHandle_t handle = 0;
	hipblasHandle_t cublasH = NULL;
	hipStream_t stream = NULL;
	hipsparseMatDescr_t descr = 0;


	hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cuda_state1 = hipSuccess;
	hipError_t cuda_state2 = hipSuccess;
	hipError_t cuda_state3 = hipSuccess;
	hipError_t cuda_state4 = hipSuccess;
	hipError_t cuda_state5 = hipSuccess;
	int *csr_gpu_x = NULL;
	int *csr_gpu_offset = NULL;
	int *csr_gpu_y = NULL;
	double *csr_gpu_value = NULL;
	double *csr_gpu_b = NULL; 
	double *csr_gpu_r = NULL;
	double *csr_gpu_d = NULL;
	double *csr_gpu_dx = NULL; 


	//allocate gpu space
	cuda_state1 = hipMalloc((void**)&csr_gpu_value, sizeof(double) * m_Hermit_R.m_NonZeros);
	cuda_state2 = hipMalloc((void**)&csr_gpu_y, sizeof(int) * m_Hermit_R.m_NonZeros);
	cuda_state3 = hipMalloc((void**)&csr_gpu_x, sizeof(int) * m_Hermit_R.m_NonZeros);
	cuda_state4 = hipMalloc((void**)&csr_gpu_b, sizeof(double) * m_Size);
	cuda_state5 = hipMalloc((void**)&csr_gpu_dx, sizeof(double) * m_Size);
	assert(cuda_state1 == hipSuccess);
	assert(cuda_state2 == hipSuccess);
	assert(cuda_state3 == hipSuccess);
	assert(cuda_state4 == hipSuccess);
	assert(cuda_state5 == hipSuccess);

	cuda_state1 = hipMalloc((void**)&csr_gpu_offset, sizeof(int) * (m_Size + 1));
	assert(cuda_state1 == hipSuccess);


	cuda_state1 = hipMalloc((void**)&csr_gpu_d, sizeof(double) * m_Size);
	cuda_state2 = hipMalloc((void**)&csr_gpu_r, sizeof(double) * m_Size);
	assert(cuda_state1 == hipSuccess);
	assert(cuda_state2 == hipSuccess);

	//copy cpu data to gpu
	cuda_state1 = hipMemcpy(csr_gpu_value, m_Hermit_R.m_Data, sizeof(double) * m_Hermit_R.m_NonZeros, hipMemcpyHostToDevice);
	cuda_state2 = hipMemcpy(csr_gpu_y, m_Hermit_R.m_Place_y, sizeof(int) * m_Hermit_R.m_NonZeros, hipMemcpyHostToDevice);
	cuda_state3 = hipMemcpy(csr_gpu_x, m_Hermit_R.m_Place_x, sizeof(int) * m_Hermit_R.m_NonZeros, hipMemcpyHostToDevice);
	cuda_state4 = hipMemcpy(csr_gpu_b, m_B_R, sizeof(double) * m_Size, hipMemcpyHostToDevice);
	assert(cuda_state1 == hipSuccess);
	assert(cuda_state2 == hipSuccess);
	assert(cuda_state3 == hipSuccess);
	assert(cuda_state4 == hipSuccess);



	//init handles
	/* initialize cusparse library */
	status = hipsparseCreate(&handle);
	assert(status == HIPSPARSE_STATUS_SUCCESS);

	/* create and setup matrix descriptor */
	status = hipsparseCreateMatDescr(&descr);
	assert(status == HIPSPARSE_STATUS_SUCCESS);

	/*cublas*/
	status_cub = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == status_cub);

	status_cub = hipblasSetStream(cublasH, stream);
	assert(HIPBLAS_STATUS_SUCCESS == status_cub);

	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	//init csr version H
	status = hipsparseXcoo2csr(handle, csr_gpu_x,
		m_Hermit_R.m_NonZeros, m_Size, csr_gpu_offset, HIPSPARSE_INDEX_BASE_ZERO);
	assert(status == HIPSPARSE_STATUS_SUCCESS);


	//init dx=0,d=b,r=b
	cuda_state1 = hipMemset(csr_gpu_dx, 0, m_Size * sizeof(double));
	assert(cuda_state1 == hipSuccess);

	cuda_state1 = hipMemcpy(csr_gpu_d, csr_gpu_b, sizeof(double) * m_Size, hipMemcpyDeviceToDevice);
	cuda_state2 = hipMemcpy(csr_gpu_r, csr_gpu_b, sizeof(double) * m_Size, hipMemcpyDeviceToDevice);
	assert(cuda_state1 == hipSuccess);
	assert(cuda_state2 == hipSuccess);

	//iteration
	for (int i = 1; i <= m_Size; i++)
	{

		//calculate ak
		double alpha = 0;
		//get alpha k
		double rk_norm = 0;//����
		double dk_norm = 0;//��ĸ
		status_cub = hipblasDdot(cublasH, m_Size,
			csr_gpu_r, 1,
			csr_gpu_r, 1,
			&rk_norm);
			assert(HIPBLAS_STATUS_SUCCESS == status_cub);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);


		double* temp = NULL; //temp = Qd
		cuda_state1 = hipMalloc((void**)&temp, sizeof(double) * m_Size);;
		assert(cuda_state1 == hipSuccess);

		cuda_state1 = hipMemset(temp, 0, sizeof(double) * m_Size);
		assert(cuda_state1 == hipSuccess);
		double temp_alpha = 1, temp_beta = 0;


		status = hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			m_Size, m_Size, m_Hermit_R.m_NonZeros, &temp_alpha,
			descr,
			csr_gpu_value,
			csr_gpu_offset, csr_gpu_y,
			csr_gpu_d, &temp_beta,
			temp);
		assert(status == HIPSPARSE_STATUS_SUCCESS);

		//test
		/*double temp_norm = 0;
		status_cub = hipblasDnrm2(cublasH, m_Size, temp, 1, &temp_norm);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);*/



		status_cub = hipblasDdot(cublasH, m_Size,
			csr_gpu_d, 1,
			temp, 1,
			&dk_norm);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);


		alpha = rk_norm / dk_norm;


		//calculate dxk
		status_cub = hipblasDaxpy(cublasH, m_Size,
			&alpha,
			csr_gpu_d, 1,
			csr_gpu_dx, 1);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);
		

		//test
		/*double dx_norm = 0;
		status_cub = hipblasDnrm2(cublasH, m_Size, csr_gpu_dx, 1, &dx_norm);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);*/


		//test
		/*double* old_r = NULL;
		hipMalloc((void**)&old_r, m_Size * sizeof(double));
		hipMemcpy(old_r, csr_gpu_r, m_Size * sizeof(double), hipMemcpyDeviceToDevice);
		*/

		//calculate rk
		double minus_alpha = 0 - alpha;
		status_cub = hipblasDaxpy(cublasH, m_Size,
			&minus_alpha,
			temp, 1,
			csr_gpu_r, 1);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);
		
		//test
		/*double kk = 0;
		status_cub = hipblasDdot(cublasH, m_Size,
			csr_gpu_r, 1,
			old_r, 1,
			&kk);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);
		*/

		double rk_plus1_norm = 0;
		status_cub = hipblasDdot(cublasH, m_Size,
			csr_gpu_r, 1,
			csr_gpu_r, 1,
			&rk_plus1_norm);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);

		if (rk_plus1_norm <= m_MaxInSolve)
		{
			hipFree(temp);
			break;
		}

		//calculate betak
		double beta = rk_plus1_norm / rk_norm;

		//calculate dk+1
		double* new_temp = NULL;
		cuda_state1 = hipMalloc((void**)&new_temp, sizeof(double) * m_Size);;
		assert(cuda_state1 == hipSuccess);

		cuda_state1 = hipMemset(new_temp,0,sizeof(double) * m_Size);
		assert(cuda_state1 == hipSuccess);


		status_cub = hipblasDaxpy(cublasH, m_Size,
			&beta,
			csr_gpu_d, 1,
			new_temp, 1);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);

		double one = 1;
		status_cub = hipblasDaxpy(cublasH, m_Size,
			&one,
			csr_gpu_r, 1,
			new_temp, 1);
		assert(HIPBLAS_STATUS_SUCCESS == status_cub);
		hipMemcpy(csr_gpu_d, new_temp, m_Size * sizeof(double), hipMemcpyDeviceToDevice);
		hipFree(temp);
		hipFree(new_temp);
		//hipFree(old_r);
	}




	//get result and end
	double max = 0;
	status_cub = hipblasDnrm2(cublasH, m_Size, csr_gpu_dx, 1, &max);
	assert(HIPBLAS_STATUS_SUCCESS == status_cub);

	hipMemcpy(m_DX_R, csr_gpu_dx, m_Size * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(csr_gpu_value);
	hipFree(csr_gpu_x);
	hipFree(csr_gpu_y);
	hipFree(csr_gpu_b);
	hipFree(csr_gpu_offset);
	hipFree(csr_gpu_dx);
	hipFree(csr_gpu_r);
	hipFree(csr_gpu_d);

	return max;
}